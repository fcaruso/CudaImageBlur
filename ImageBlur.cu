#include "hip/hip_runtime.h"
#include <iostream>

// CUDA utilities and system includes
#include <hip/hip_runtime.h>

// Helper functions
#include <helper_functions.h>  // CUDA SDK Helper functions
#include <hip/hip_runtime_api.h>       // CUDA device initialization helper functions
#include <helper_image.h>

#include <hip/hip_runtime_api.h>

char *image_filename = "./data/lena.pgm";
unsigned int width, height;
unsigned char *h_img  = NULL;
unsigned char *d_img  = NULL;

#define BLOCK_WIDTH		64
#define BLOCK_HEIGHT	16

#ifndef __CUDA_ARCH__
#ifdef WIN32
#pragma message ("__CUDA_ARCH__ undefined")
#else
#warning "__CUDA_ARCH__ undefined"
#endif
#endif 


//////////////////////////////////////
/// Radial blur using global memory
//////////////////////////////////////
template<unsigned short RADIUS >
__global__ void kRadialBlur( unsigned char* img, unsigned width, unsigned height, size_t pitch)
{
	__shared__ unsigned char sh[BLOCK_HEIGHT + 2*RADIUS][BLOCK_WIDTH + 2*RADIUS];

	int g_x = blockDim.x*blockIdx.x + threadIdx.x;
	int g_y = blockDim.y*blockIdx.y + threadIdx.y;

	int pid_x = threadIdx.x + RADIUS;
	int pid_y = threadIdx.y + RADIUS;
	
	///////////////////////
	// gather into shared memory
	///////////////////////
	sh[pid_y][pid_x] = img[ g_y*pitch + g_x];

	// halo 
	if ( ( threadIdx.x < RADIUS ) && ( g_x  >= RADIUS ) )
	{
		sh[pid_y][pid_x - RADIUS] = img[ g_y*pitch + g_x - RADIUS];

		if ( ( threadIdx.y < RADIUS ) && ( g_y >= RADIUS ) )
		{
			sh[pid_y - RADIUS][pid_x - RADIUS] = img[ (g_y - RADIUS)*pitch + g_x - RADIUS];
		}
		if ( ( threadIdx.y > (BLOCK_HEIGHT -1 - RADIUS) ) )
		{
			sh[pid_y + RADIUS][pid_x - RADIUS] = img[ (g_y + RADIUS)*pitch + g_x - RADIUS];
		}
	}
	if ( ( threadIdx.x > ( BLOCK_WIDTH -1 - RADIUS ) ) && ( g_x < ( width - RADIUS ) ) )
	{
		sh[pid_y][pid_x + RADIUS ] = img[ g_y*pitch + g_x + RADIUS];

		if ( ( threadIdx.y < RADIUS ) && ( g_y > RADIUS ) )
		{
			sh[pid_y - RADIUS][pid_x + RADIUS] = img[ (g_y - RADIUS)*pitch + g_x + RADIUS];
		}
		if ( (threadIdx.y > (BLOCK_HEIGHT -1 - RADIUS ) ) && ( g_y < ( height - RADIUS ) ) )
		{
			sh[pid_y + RADIUS][pid_x + RADIUS] = img[ (g_y + RADIUS)*pitch + g_x + RADIUS];
		}
	}

	if ( ( threadIdx.y < RADIUS ) && ( g_y >= RADIUS ) )
	{
		sh[pid_y - RADIUS][pid_x] = img[ (g_y - RADIUS)*pitch + g_x];
	}
	if ( ( threadIdx.y > ( BLOCK_HEIGHT -1 - RADIUS ) ) && ( g_y < ( height - RADIUS ) ) )
	{
		sh[pid_y + RADIUS][pid_x] = img[ ( g_y + RADIUS)*pitch + g_x ];
	}

	__syncthreads();

	//////////////////////
	// compute the blurred value
	//////////////////////

	unsigned val = 0;
	unsigned k = 0;
	for (int i=-RADIUS; i<= RADIUS; i++ )
		for ( int j=-RADIUS; j<=RADIUS ; j++ )
		{
			if ( ( ( g_x + j ) < 0 ) || ( ( g_x + j ) > ( width - 1) ) )
				continue;
			if ( ( ( g_y + i ) < 0 ) || ( ( g_y + i ) > ( height - 1) ) )
				continue;
			val += sh[pid_y + i][pid_x + j];
			k++;
		}

	val /= k;

	////////////////////
	// write into global memory
	///////////////

	img[ g_y*pitch + g_x ] = (unsigned char) val;
			
}

//////////////////////////////////////
/// Radial blur using texture memory
//////////////////////////////////////

template<unsigned short RADIUS>
__global__ void kRadialBlur( unsigned char* img, hipTextureObject_t tex,
							unsigned width, unsigned height, size_t pitch)
{
	__shared__unsigned char sh[BLOCK_HEIGHT + 2*RADIUS][BLOCK_WIDTH + 2*RADIUS];

	int g_x = blockDim.x*blockIdx.x + threadIdx.x;
	int g_y = blockDim.y*blockIdx.y + threadIdx.y;

	int pid_x = threadIdx.x + RADIUS;
	int pid_y = threadIdx.y + RADIUS;
	
	///////////////////////
	// gather into shared memory
	///////////////////////
	sh[pid_y][pid_x] = tex2D<unsigned char>(tex, g_x, g_y);
	
	// halo 
	if ( ( threadIdx.x < RADIUS ) && ( g_x  >= RADIUS ) )
	{
		sh[pid_y][pid_x - RADIUS] = tex2D<unsigned char>(tex, g_x - RADIUS , g_y);

		if ( ( threadIdx.y < RADIUS ) && ( g_y >= RADIUS ) )
		{
			sh[pid_y - RADIUS][pid_x - RADIUS] = tex2D<unsigned char>(tex, g_x , g_y - RADIUS);
		}
		if ( ( threadIdx.y > (BLOCK_HEIGHT -1 - RADIUS) ) )
		{
			sh[pid_y + RADIUS][pid_x - RADIUS] = tex2D<unsigned char>(tex, g_x - RADIUS, g_y - RADIUS);
		}
	}
	if ( ( threadIdx.x > ( BLOCK_WIDTH -1 - RADIUS ) ) && ( g_x < ( width - RADIUS ) ) )
	{
		sh[pid_y][pid_x + RADIUS ] = tex2D<T>(tex, g_x + RADIUS, g_y );

		if ( ( threadIdx.y < RADIUS ) && ( g_y > RADIUS ) )
		{
			sh[pid_y - RADIUS][pid_x + RADIUS] = tex2D<unsigned char>(tex, g_x + RADIUS, g_y - RADIUS);
		}
		if ( (threadIdx.y > (BLOCK_HEIGHT -1 - RADIUS ) ) && ( g_y < ( height - RADIUS ) ) )
		{
			sh[pid_y + RADIUS][pid_x + RADIUS] = tex2D<unsigned char>(tex, g_x + RADIUS, g_y + RADIUS);
		}
	}

	if ( ( threadIdx.y < RADIUS ) && ( g_y >= RADIUS ) )
	{
		sh[pid_y - RADIUS][pid_x] = tex2D<unsigned char>(tex, g_x , g_y - RADIUS);
	}
	if ( ( threadIdx.y > ( BLOCK_HEIGHT -1 - RADIUS ) ) && ( g_y < ( height - RADIUS ) ) )
	{
		sh[pid_y + RADIUS][pid_x] = tex2D<unsigned char>(tex, g_x , g_y + RADIUS);
	}

	__syncthreads();

	//////////////////////
	// compute the blurred value
	//////////////////////

	unsigned val = 0;
	unsigned k = 0;
	for (int i=-RADIUS; i<= RADIUS; i++ )
		for ( int j=-RADIUS; j<=RADIUS ; j++ )
		{
			if ( ( ( g_x + j ) < 0 ) || ( ( g_x + j ) > ( width - 1) ) )
				continue;
			if ( ( ( g_y + i ) < 0 ) || ( ( g_y + i ) > ( height - 1) ) )
				continue;
			val += sh[pid_y + i][pid_x + j];
			k++;
		}

	val /= k;

	////////////////////
	// write into global memory
	///////////////

	img[ g_y*pitch + g_x ] = (unsigned char) val;
			
}

int main(int argc, char* argv[])
{
	hipSetDevice(0);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);

	hipProfilerStart();
	hipError_t err;
    // load image (needed so we can get the width and height before we create the window
	sdkLoadPGM(image_filename, (unsigned char **) &h_img, &width, &height);
	printf("width: %d \t height: %d \n", width, height);

	// fill GPU  memory
	unsigned char* d_img = NULL;
	size_t pitch;
	hipMallocPitch( (void**) &d_img, &pitch, width*sizeof(unsigned char), height );
	hipMemcpy2D( d_img, pitch*sizeof(unsigned char), 
			h_img, width*sizeof(unsigned char), width*sizeof(unsigned char), height, 
			hipMemcpyHostToDevice );

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned char>();
	hipArray* cuArray;
	hipMallocArray(&cuArray, &channelDesc, width, height);
	hipMemcpyToArray(cuArray, 0, 0, h_img, 
		width*height*sizeof(unsigned char),
		hipMemcpyHostToDevice );

	hipResourceDesc resDesc;
	memset( &resDesc, 0, sizeof(resDesc) );
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = cuArray;

	hipTextureDesc texDesc;
	memset( &texDesc, 0, sizeof( texDesc ) );
	texDesc.addressMode[0] = hipAddressModeClamp;
	texDesc.addressMode[1] = hipAddressModeClamp;
	texDesc.filterMode = hipFilterModePoint;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = false;

	hipTextureObject_t texObj = 0;
	hipCreateTextureObject( &texObj, &resDesc, &texDesc, NULL );

	// create vars for timing
	hipEvent_t startEvent, stopEvent;
	err = hipEventCreate(&startEvent, 0);
	assert( err == hipSuccess );
	err = hipEventCreate(&stopEvent, 0);
	assert( err == hipSuccess );
	float elapsedTime;

	// process image
	dim3 dGrid(width / BLOCK_WIDTH, height / BLOCK_HEIGHT);
	dim3 dBlock(BLOCK_WIDTH, BLOCK_HEIGHT);
	
	// execution of the version using global memory
	hipEventRecord(startEvent);
	kRadialBlur<4> <<< dGrid, dBlock >>> (d_img, width, height, pitch );
	hipDeviceSynchronize();
	hipEventRecord(stopEvent);
	hipEventSynchronize( stopEvent );
	hipEventElapsedTime( &elapsedTime, startEvent, stopEvent);

	printf("elapsed time of version using global memory: %f\n", elapsedTime );

	// execution of the version using texture memory
	if ( deviceProp.major >= 3 ) // Texture objects are supported from arch 3.X
	{
		hipEventRecord(startEvent);
		kRadialBlur<4> <<< dGrid, dBlock >>> (d_img, texObj, width, height, pitch );
		hipDeviceSynchronize();
		hipEventRecord(stopEvent);
		hipEventSynchronize( stopEvent );
		hipEventElapsedTime( &elapsedTime, startEvent, stopEvent);

		printf("elapsed time of version using texture memory: %f\n", elapsedTime );

	}
	else
	{
        printf("CUDA Texture Object requires a GPU with compute capability "
               "3.0 or later\n");
	}
	// save image
	hipMemcpy2D( h_img, width*sizeof(unsigned char), 
		d_img, pitch*sizeof(unsigned char), width*sizeof(unsigned char), height,
		hipMemcpyDeviceToHost );
	sdkSavePGM("./data/blurred_tex.ppm", h_img, width, height );

	// free memory
	hipDestroyTextureObject(texObj);
	hipFreeArray(cuArray);
	hipFree(d_img);
	hipProfilerStop();
	hipDeviceReset();
	free(h_img);

	return 0;
}